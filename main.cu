#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// #include "modules/drug_sim.hpp"
#include "modules/glob_funct.hpp"
#include "modules/glob_type.hpp"
#include "modules/gpu.cuh"

#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <ctime>
#include <iostream>
#include <math.h>
#include <vector>

#define ENOUGH ((CHAR_BIT * sizeof(int) - 1) / 3 + 2)
char buffer[255];
double ic50[14*2000]; //temporary
unsigned int datapoint_size = 7000;

clock_t START_TIMER;

clock_t tic();
void toc(clock_t start = START_TIMER);

clock_t tic()
{
    return START_TIMER = clock();
}

void toc(clock_t start)
{
    std::cout
        << "Elapsed time: "
        << (clock() - start) / (double)CLOCKS_PER_SEC << "s"
        << std::endl;
}

// since installing MPI in Windows
// is quite a hassle, don't bother
// to use it in Windows.
// #ifndef _WIN32
// 	#include <mpi.h>
// #endif

// constants to avoid magic values
// static const char *RESULT_FOLDER_PATH = "result";
// static const double CONTROL_CONC = 0.;


// get the IC50 data from file
drug_t get_IC50_data_from_file(const char* file_name);
// return error and message based on the IC50 data
int check_IC50_content(const drug_t* ic50, const param_t* p_param);

// define MPI data structure for qinward_t to be broadcasted
// #ifndef _WIN32
// MPI_Datatype create_mpi_qinward_t();
// #endif

// drug_t get_IC50_data_from_file(const char* file_name)
// {
//   FILE *fp_drugs;
//   drug_t ic50;
//   char *token, buffer[255];
//   row_data temp_array;
//   unsigned short idx;

//   if( (fp_drugs = fopen(file_name, "r")) == NULL){
//     printf("Cannot open file %s in %s at rank %d\n",
//       file_name, mympi::host_name, mympi::rank);
//     return ic50;
//   }

//   fgets(buffer, sizeof(buffer), fp_drugs); // skip header
//   while( fgets(buffer, sizeof(buffer), fp_drugs) != NULL )
//   { // begin line reading
//     token = strtok( buffer, "," );
//     idx = 0;
//     while( token != NULL )
//     { // begin data tokenizing
//       temp_array.data[idx++] = strtod(token, NULL);
//       token = strtok(NULL, ",");
//     } // end data tokenizing
//     ic50.push_back(temp_array);
//   } // end line reading

//   fclose(fp_drugs);
//   return ic50;
// }

int get_IC50_data_from_file(const char* file_name, double *ic50)
{
    /*
    a host function to take all samples from the file, assuming each sample has 14 features.

    it takes the file name, and an ic50 (already declared in 1D, everything become 1D)
    as a note, the data will be stored in 1D array, means this functions applies flatten.

    it returns 'how many samples were detected?' in integer.
    */
  FILE *fp_drugs;
//   drug_t ic50;
  char *token;
  
  unsigned short idx;

  if( (fp_drugs = fopen(file_name, "r")) == NULL){
    printf("Cannot open file %s\n",
      file_name);
    return 0;
  }
  idx = 0;
  int sample_size = 0;
  fgets(buffer, sizeof(buffer), fp_drugs); // skip header
  while( fgets(buffer, sizeof(buffer), fp_drugs) != NULL )
  { // begin line reading
    token = strtok( buffer, "," );
    while( token != NULL )
    { // begin data tokenizing
      ic50[idx++] = strtod(token, NULL);
      token = strtok(NULL, ",");
    } // end data tokenizing
    sample_size++;
  } // end line reading

  fclose(fp_drugs);
  return sample_size;
}


int check_IC50_content(const drug_t* ic50, const param_t* p_param)
{
	if(ic50->size() == 0){
		printf("Something problem with the IC50 file!\n");
		return 1;
	}
	else if(ic50->size() > 2000){
		printf( "Too much input! Maximum sample data is 2000!\n");
		return 2;
	}
	else if(p_param->pace_max < 750 && p_param->pace_max > 1000){
		printf("Make sure the maximum pace is around 750 to 1000!\n");
		return 3;
	}
	// else if(mympi::size > ic50->size()){
	// 	printf("%s\n%s\n",
  //               "Overflow of MPI Process!",
  //               "Make sure MPI Size is less than or equal the number of sample");
	// 	return 4;
	// }
	else{
		return 0;
	}
}

int main(int argc, char **argv)
{
	// enable real-time output in stdout
	setvbuf( stdout, NULL, _IONBF, 0 );
	
// #ifndef _WIN32
// 	MPI_Init( &argc, &argv );
// 	MPI_Comm_size( MPI_COMM_WORLD, &mympi::size );
// 	MPI_Comm_rank( MPI_COMM_WORLD, &mympi::rank );
// 	MPI_Get_processor_name(mympi::host_name, &mympi::host_name_len);
// #else
// 	mympi::size = 1;
// 	mympi::rank = 0;
// 	snprintf(mympi::host_name,sizeof(mympi::host_name),"%s","host");
// 	mympi::host_name_len = 4;
// #endif

// NEW CODE STARTS HERE //
    // mycuda *thread_id;
    // hipMalloc(&thread_id, sizeof(mycuda));

    double *d_ic50;
    double *d_ALGEBRAIC;
    double *d_CONSTANTS;
    double *d_RATES;
    double *d_STATES;

    double *time;
    double *dt;
    double *states;
    double *ical;
    double *inal;
    double *cai_result;
    double *ina;
    double *ito;
    double *ikr;
    double *iks;
    double *ik1;

    static const int CALCIUM_SCALING = 1000000;
    static const int CURRENT_SCALING = 1000;

    // input variables for cell simulation
    param_t *p_param, *d_p_param;
	  p_param = new param_t();
  	p_param->init();

    // p_param->show_val();

    int num_of_constants = 146;
    int num_of_states = 41;
    int num_of_algebraic = 199;
    int num_of_rates = 41;

    snprintf(buffer, sizeof(buffer),
      "./drugs/bepridil/IC50_samples.csv"
      // "./drugs/bepridil/IC50_optimal.csv"
      // "./IC50_samples.csv"
      );
    int sample_size = get_IC50_data_from_file(buffer, ic50);
    if(sample_size == 0)
        printf("Something problem with the IC50 file!\n");
    // else if(sample_size > 2000)
    //     printf("Too much input! Maximum sample data is 2000!\n");
    printf("Sample size: %d\n",sample_size);
   
    printf("preparing GPU memory space \n");
    hipMalloc(&d_ALGEBRAIC, num_of_algebraic * sample_size * sizeof(double));
    hipMalloc(&d_CONSTANTS, num_of_constants * sample_size * sizeof(double));
    hipMalloc(&d_RATES, num_of_rates * sample_size * sizeof(double));
    hipMalloc(&d_STATES, num_of_states * sample_size * sizeof(double));
    hipMalloc(&d_p_param,  sizeof(param_t));
    // prep for 1 cycle plus a bit (700 * sample_size)
    hipMalloc(&time, sample_size * datapoint_size * sizeof(double)); 
    hipMalloc(&dt, sample_size * datapoint_size * sizeof(double)); 
    hipMalloc(&states, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&ical, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&inal, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&cai_result, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&ina, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&ito, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&ikr, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&iks, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&ik1, sample_size * datapoint_size * sizeof(double));

    printf("Copying sample files to GPU memory space \n");
    hipMalloc(&d_ic50, sample_size * 14 * sizeof(double));
    hipMemcpy(d_ic50, ic50, sample_size * 14 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_p_param, p_param, sizeof(param_t), hipMemcpyHostToDevice);

    // // Get the maximum number of active blocks per multiprocessor
    // hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, do_drug_sim_analytical, threadsPerBlock);

    // // Calculate the total number of blocks
    // int numTotalBlocks = numBlocks * cudaDeviceGetMultiprocessorCount();

    tic();
    printf("Timer started, doing simulation.... \n");
    int thread = 100;
    int block = int(ceil(sample_size/thread));
    // int block = (sample_size + thread - 1) / thread;

    printf("Sample size: %d\n",sample_size);
    printf("\n   Configuration: \n\n\tblock\t||\tthread\n---------------------------------------\n  \t%d\t||\t%d\n\n\n", block,thread);
    // initscr();
    // printf("[____________________________________________________________________________________________________]  0.00 %% \n");

    kernel_DrugSimulation<<<block,thread>>>(d_ic50, d_CONSTANTS, d_STATES, d_RATES, d_ALGEBRAIC, 
                                              time, states, dt, cai_result,
                                              ina, inal, 
                                              ical, ito,
                                              ikr, iks, 
                                              ik1,
                                              sample_size,
                                              d_p_param);
                                      //block per grid, threads per block
    // endwin();
    hipDeviceSynchronize();
    

    printf("allocating memory for computation result in the CPU, malloc style \n");
    double *h_states,*h_time,*h_dt,*h_ical,*h_inal,*h_cai_result,*h_ina,*h_ito,*h_ikr,*h_iks,*h_ik1;
;

    h_states = (double *)malloc(datapoint_size * sample_size * sizeof(double));
    printf("...allocated for STATES, \n");
    h_time = (double *)malloc(datapoint_size * sample_size * sizeof(double));
    printf("...allocated for time, \n");
    h_dt = (double *)malloc(datapoint_size * sample_size * sizeof(double));
    printf("...allocated for dt, \n");
    h_cai_result= (double *)malloc(datapoint_size * sample_size * sizeof(double));
    printf("...allocated for Cai, \n");
     h_ina= (double *)malloc(datapoint_size * sample_size * sizeof(double));
    printf("...allocated for iNa, \n");
     h_ito= (double *)malloc(datapoint_size * sample_size * sizeof(double));
    printf("...allocated for ito, \n");
     h_ikr= (double *)malloc(datapoint_size * sample_size * sizeof(double));
    printf("...allocated for ikr, \n");
     h_iks= (double *)malloc(datapoint_size * sample_size * sizeof(double));
    printf("...allocated for iks, \n");
     h_ik1= (double *)malloc(datapoint_size * sample_size * sizeof(double));
    printf("...allocated for ik1, \n");
     h_ical= (double *)malloc(datapoint_size * sample_size * sizeof(double));
    printf("...allocated for ICaL, \n");
    h_inal = (double *)malloc(datapoint_size * sample_size * sizeof(double));
    printf("...allocating for INaL, all set!\n");

    ////// copy the data back to CPU, and write them into file ////////
    printf("copying the data back to the CPU \n");
    hipMemcpy(h_states, states, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_time, time, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_dt, dt, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_ical, ical, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_inal, inal, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_cai_result, cai_result, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_ina, ina, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_ito, ito, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_ikr, ikr, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_iks, iks, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_ik1, ik1, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
    

    FILE *writer;

    printf("writing to file... \n");
    // sample loop
    for (int sample_id = 0; sample_id<sample_size; sample_id++){
      
      char sample_str[ENOUGH];
      char filename[150] = "./result/sober/";
      sprintf(sample_str, "%d", sample_id);
      strcat(filename,sample_str);
      strcat(filename,".csv");

      writer = fopen(filename,"w");
      fprintf(writer, "Time","Vm","dVm/dt","Cai(x1.000.000)(milliM->picoM)",
         "INa(x1.000)(microA->picoA)","INaL(x1.000)(microA->picoA)","ICaL(x1.000)(microA->picoA)",
         "IKs(x1.000)(microA->picoA)","IKr(x1.000)(microA->picoA)","IK1(x1.000)(microA->picoA)",
         "Ito(x1.000)(microA->picoA)"); 
      for (int datapoint = 0; datapoint<datapoint_size; datapoint++){
       // if (h_time[ sample_id + (datapoint * sample_size)] == 0.0) {continue;}
        fprintf(writer,"%s,%s,%s,%s,%s,%s,%s,%s,%s,%s,%s\n",
        h_time[ sample_id + (datapoint * sample_size)],
        h_states[ sample_id + (datapoint * sample_size)],
        h_dt[ sample_id + (datapoint * sample_size)],
        h_cai_result[ sample_id + (datapoint * sample_size)]*CALCIUM_SCALING, 
        
        h_ina[ sample_id + (datapoint * sample_size)]*CURRENT_SCALING, 
        h_inal[ sample_id + (datapoint * sample_size)]*CURRENT_SCALING, 

        h_ical[ sample_id + (datapoint * sample_size)]*CURRENT_SCALING,
        h_ito[ sample_id + (datapoint * sample_size)]*CURRENT_SCALING,  

        h_ikr[ sample_id + (datapoint * sample_size)]*CURRENT_SCALING, 
        h_iks[ sample_id + (datapoint * sample_size)]*CURRENT_SCALING, 

        h_ik1[ sample_id + (datapoint * sample_size)]*CURRENT_SCALING
        );
      }
      fclose(writer);
    }
    toc();
    
    return 0;
	
}
